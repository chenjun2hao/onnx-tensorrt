#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "ResizeBilinear.hpp"
#include <hip/hip_fp16.h>
#include <cassert>

// TODO: Move this to a common header
inline bool is_CHW(nvinfer1::Dims const& dims) {
  return (dims.nbDims == 3 &&
          dims.type[0] == nvinfer1::DimensionType::kCHANNEL &&
          dims.type[1] == nvinfer1::DimensionType::kSPATIAL &&
          dims.type[2] == nvinfer1::DimensionType::kSPATIAL);
}

nvinfer1::Dims ResizeBilinearPlugin::getOutputDimensions(int index,
                                                        const nvinfer1::Dims *inputDims,
                                                        int nbInputs) {
  assert(nbInputs == 1);
  nvinfer1::Dims const& input = inputDims[0];
  assert(is_CHW(input));
  assert(_ndims == 2);
  assert(index == 0);
  nvinfer1::Dims output;
  output.nbDims = input.nbDims;
  int s = 0;
  for( int d=0; d<input.nbDims; ++d ) {
    output.type[d] = input.type[d];
    if( input.type[d] == nvinfer1::DimensionType::kSPATIAL ) {
      output.d[d] = int(input.d[d] * _scale[s++]);
    } else {
      output.d[d] = input.d[d];
    }
  }
  return output;
}

int ResizeBilinearPlugin::initialize() {
  _output_dims = this->getOutputDimensions(0, &this->getInputDims(0), 1);
  assert(is_CHW(this->getInputDims(0)));
  assert(is_CHW(_output_dims));
  assert(_ndims == 2);
  return 0;
}


template <typename Data>
__global__
void resize_bilinear_kernel_2d(int nbatch,
                              float2 scale,
                              int2 isize,
                              int2 osize,
                              Data const* idata, int istride, int ibatchstride,
                              Data*       odata, int ostride, int obatchstride) {
  int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  int y0 = threadIdx.y + blockIdx.y * blockDim.y;
  int z0 = blockIdx.z;
  for( int batch=z0; batch<nbatch; batch+=gridDim.z ) {
    for( int oy=y0; oy<osize.y; oy+=blockDim.y*gridDim.y ) {
      for( int ox=x0; ox<osize.x; ox+=blockDim.x*gridDim.x ) {
        float ix = float(ox) / scale.x;
        float iy = float(oy) / scale.y;
        int w_low = ((int)ix < isize.x) ? (int)ix : (int)(isize.x - 1);
        int h_low = ((int)iy < isize.y) ? (int)iy : (int)(isize.y - 1);
        int w_high = w_low + 1 < isize.x ? w_low + 1 : (int)(isize.x - 1);
        int h_high = h_low + 1 < isize.y ? h_low + 1 : (int)(isize.y - 1);
        // int w_low = int(ix);
        // int h_low = int(iy);
        // int w_high = w_low + 1;
        // int h_high = h_low + 1;
        float lw = ix - w_low, lh = iy - h_low;
        float hw = 1 - lw,    hh = 1 - lh;
        Data W1 = (Data)(hh * hw), W2 = (Data)(hh * lw), W3 = (Data)(lh * hw), W4 = (Data)(lh * lw);
        Data v1 = idata[batch * ibatchstride + h_low * istride + w_low];
        Data v2 = idata[batch * ibatchstride + h_low * istride + w_high];
        Data v3 = idata[batch * ibatchstride + h_high * istride + w_low];
        Data v4 = idata[batch * ibatchstride + h_high * istride + w_high];
        odata[batch * obatchstride + oy * ostride + ox] = Data(W1 * v1) + Data(W2 * v2) + Data(W3 * v3) + Data(W4 * v4);
    
      }
    }
  }
}

int ResizeBilinearPlugin::enqueue(int batchSize,
                                 const void *const *inputs, void **outputs,
                                 void *workspace, hipStream_t stream) {
  auto const& input_dims = this->getInputDims(0);
  int nchan = input_dims.d[0];
  switch( _ndims ) {
  case 2: {
    float2 scale = {_scale[1], _scale[0]};
    int2 isize = {input_dims.d[2], input_dims.d[1]};
    int2 osize = {_output_dims.d[2], _output_dims.d[1]};
    int istride =   input_dims.d[2];
    int ostride = _output_dims.d[2];
    int ibatchstride =   input_dims.d[1] * istride;
    int obatchstride = _output_dims.d[1] * ostride;
    dim3 block(16, 16);
    dim3 grid((osize.x - 1) / block.x + 1,
              (osize.y - 1) / block.y + 1,
              std::min(batchSize * nchan, 65535));
    if (getDataType()==nvinfer1::DataType::kFLOAT) {				
      resize_bilinear_kernel_2d<<<grid, block, 0, stream>>>
        (batchSize * nchan, scale, isize, osize,
         static_cast<float const*>( inputs[0]), istride, ibatchstride,
         static_cast<float*      >(outputs[0]), ostride, obatchstride);
    } else {
      resize_bilinear_kernel_2d<<<grid, block, 0, stream>>>
        (batchSize * nchan, scale, isize, osize,
         static_cast<__half const*>( inputs[0]), istride, ibatchstride,
         static_cast<__half*      >(outputs[0]), ostride, obatchstride);
    }
    return hipGetLastError() != hipSuccess;
  }
  default: return -1;
  }
}
